#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/copy.h>
#include <thrust/device_ptr.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans)                                                    \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

  SceneName sceneName;

  int numCircles;
  float *position;
  float *velocity;
  float *color;
  float *radius;

  int imageWidth;
  int imageHeight;
  float *imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int cuConstNoiseYPermutationTable[256];
__constant__ int cuConstNoiseXPermutationTable[256];
__constant__ float cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float cuConstColorRamp[COLOR_MAP_SIZE][3];

// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "circleBoxTest.cu_inl"
#include "lookupColor.cu_inl"
#include "noiseCuda.cu_inl"

// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

  int imageX = blockIdx.x * blockDim.x + threadIdx.x;
  int imageY = blockIdx.y * blockDim.y + threadIdx.y;

  int width = cuConstRendererParams.imageWidth;
  int height = cuConstRendererParams.imageHeight;

  if (imageX >= width || imageY >= height)
    return;

  int offset = 4 * (imageY * width + imageX);
  float shade = .4f + .45f * static_cast<float>(height - imageY) / height;
  float4 value = make_float4(shade, shade, shade, 1.f);

  // write to global memory: As an optimization, I use a float4
  // store, that results in more efficient code than if I coded this
  // up as four seperate fp32 stores.
  *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

  int imageX = blockIdx.x * blockDim.x + threadIdx.x;
  int imageY = blockIdx.y * blockDim.y + threadIdx.y;

  int width = cuConstRendererParams.imageWidth;
  int height = cuConstRendererParams.imageHeight;

  if (imageX >= width || imageY >= height)
    return;

  int offset = 4 * (imageY * width + imageX);
  float4 value = make_float4(r, g, b, a);

  // write to global memory: As an optimization, I use a float4
  // store, that results in more efficient code than if I coded this
  // up as four seperate fp32 stores.
  *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
//
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
  const float dt = 1.f / 60.f;
  const float pi = 3.14159;
  const float maxDist = 0.25f;

  float *velocity = cuConstRendererParams.velocity;
  float *position = cuConstRendererParams.position;
  float *radius = cuConstRendererParams.radius;

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= cuConstRendererParams.numCircles)
    return;

  if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update
    return;
  }

  // determine the fire-work center/spark indices
  int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
  int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

  int index3i = 3 * fIdx;
  int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
  int index3j = 3 * sIdx;

  float cx = position[index3i];
  float cy = position[index3i + 1];

  // update position
  position[index3j] += velocity[index3j] * dt;
  position[index3j + 1] += velocity[index3j + 1] * dt;

  // fire-work sparks
  float sx = position[index3j];
  float sy = position[index3j + 1];

  // compute vector from firework-spark
  float cxsx = sx - cx;
  float cysy = sy - cy;

  // compute distance from fire-work
  float dist = sqrt(cxsx * cxsx + cysy * cysy);
  if (dist > maxDist) { // restore to starting position
    // random starting position on fire-work's rim
    float angle = (sfIdx * 2 * pi) / NUM_SPARKS;
    float sinA = sin(angle);
    float cosA = cos(angle);
    float x = cosA * radius[fIdx];
    float y = sinA * radius[fIdx];

    position[index3j] = position[index3i] + x;
    position[index3j + 1] = position[index3i + 1] + y;
    position[index3j + 2] = 0.0f;

    // travel scaled unit length
    velocity[index3j] = cosA / 5.0;
    velocity[index3j + 1] = sinA / 5.0;
    velocity[index3j + 2] = 0.0f;
  }
}

// kernelAdvanceHypnosis
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= cuConstRendererParams.numCircles)
    return;

  float *radius = cuConstRendererParams.radius;

  float cutOff = 0.5f;
  // place circle back in center after reaching threshold radisus
  if (radius[index] > cutOff) {
    radius[index] = 0.02f;
  } else {
    radius[index] += 0.01f;
  }
}

// kernelAdvanceBouncingBalls
//
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() {
  const float dt = 1.f / 60.f;
  const float kGravity = -2.8f; // sorry Newton
  const float kDragCoeff = -0.8f;
  const float epsilon = 0.001f;

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= cuConstRendererParams.numCircles)
    return;

  float *velocity = cuConstRendererParams.velocity;
  float *position = cuConstRendererParams.position;

  int index3 = 3 * index;
  // reverse velocity if center position < 0
  float oldVelocity = velocity[index3 + 1];
  float oldPosition = position[index3 + 1];

  if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition
    return;
  }

  if (position[index3 + 1] < 0 && oldVelocity < 0.f) { // bounce ball
    velocity[index3 + 1] *= kDragCoeff;
  }

  // update velocity: v = u + at (only along y-axis)
  velocity[index3 + 1] += kGravity * dt;

  // update positions (only along y-axis)
  position[index3 + 1] += velocity[index3 + 1] * dt;

  if (fabsf(velocity[index3 + 1] - oldVelocity) < epsilon &&
      oldPosition < 0.0f &&
      fabsf(position[index3 + 1] - oldPosition) < epsilon) { // stop ball
    velocity[index3 + 1] = 0.f;
    position[index3 + 1] = 0.f;
  }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= cuConstRendererParams.numCircles)
    return;

  const float dt = 1.f / 60.f;
  const float kGravity = -1.8f; // sorry Newton
  const float kDragCoeff = 2.f;

  int index3 = 3 * index;

  float *positionPtr = &cuConstRendererParams.position[index3];
  float *velocityPtr = &cuConstRendererParams.velocity[index3];

  // loads from global memory
  float3 position = *((float3 *)positionPtr);
  float3 velocity = *((float3 *)velocityPtr);

  // hack to make farther circles move more slowly, giving the
  // illusion of parallax
  float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

  // add some noise to the motion to make the snow flutter
  float3 noiseInput;
  noiseInput.x = 10.f * position.x;
  noiseInput.y = 10.f * position.y;
  noiseInput.z = 255.f * position.z;
  float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
  noiseForce.x *= 7.5f;
  noiseForce.y *= 5.f;

  // drag
  float2 dragForce;
  dragForce.x = -1.f * kDragCoeff * velocity.x;
  dragForce.y = -1.f * kDragCoeff * velocity.y;

  // update positions
  position.x += velocity.x * dt;
  position.y += velocity.y * dt;

  // update velocities
  velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
  velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

  float radius = cuConstRendererParams.radius[index];

  // if the snowflake has moved off the left, right or bottom of
  // the screen, place it back at the top and give it a
  // pseudorandom x position and velocity.
  if ((position.y + radius < 0.f) || (position.x + radius) < -0.f ||
      (position.x - radius) > 1.f) {
    noiseInput.x = 255.f * position.x;
    noiseInput.y = 255.f * position.y;
    noiseInput.z = 255.f * position.z;
    noiseForce = cudaVec2CellNoise(noiseInput, index);

    position.x = .5f + .5f * noiseForce.x;
    position.y = 1.35f + radius;

    // restart from 0 vertical velocity.  Choose a
    // pseudo-random horizontal velocity.
    velocity.x = 2.f * noiseForce.y;
    velocity.y = 0.f;
  }

  // store updated positions and velocities to global memory
  *((float3 *)positionPtr) = position;
  *((float3 *)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void shadePixel(int circleIndex, float2 pixelCenter,
                                      float3 p, float4 *imagePtr) {

  float diffX = p.x - pixelCenter.x;
  float diffY = p.y - pixelCenter.y;
  float pixelDist = diffX * diffX + diffY * diffY;

  float rad = cuConstRendererParams.radius[circleIndex];
  ;
  float maxDist = rad * rad;

  // circle does not contribute to the image
  if (pixelDist > maxDist)
    return;

  float3 rgb;
  float alpha;

  // there is a non-zero contribution.  Now compute the shading value

  // suggestion: This conditional is in the inner loop.  Although it
  // will evaluate the same for all threads, there is overhead in
  // setting up the lane masks etc to implement the conditional.  It
  // would be wise to perform this logic outside of the loop next in
  // kernelRenderCircles.  (If feeling good about yourself, you
  // could use some specialized template magic).
  if (cuConstRendererParams.sceneName == SNOWFLAKES ||
      cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

    const float kCircleMaxAlpha = .5f;
    const float falloffScale = 4.f;

    float normPixelDist = sqrt(pixelDist) / rad;
    rgb = lookupColor(normPixelDist);

    float maxAlpha = .6f + .4f * (1.f - p.z);
    maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f),
                                       0.f); // kCircleMaxAlpha * clamped value
    alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

  } else {
    // simple: each circle has an assigned color
    int index3 = 3 * circleIndex;
    rgb = *(float3 *)&(cuConstRendererParams.color[index3]);
    alpha = .5f;
  }

  float oneMinusAlpha = 1.f - alpha;

  // BEGIN SHOULD-BE-ATOMIC REGION
  // global memory read

  float4 existingColor = *imagePtr;
  float4 newColor;
  newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
  newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
  newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
  newColor.w = alpha + existingColor.w;

  // global memory write
  *imagePtr = newColor;

  // END SHOULD-BE-ATOMIC REGION
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a pixel.
__global__ void kernelRenderCircles(int squareSize, int *numCircles,
                                    int *indexOffsets, int *circleIndices) {
  int pixelX = blockIdx.x * blockDim.x + threadIdx.x;
  int pixelY = blockIdx.y * blockDim.y + threadIdx.y;

  short imageWidth = cuConstRendererParams.imageWidth;
  short imageHeight = cuConstRendererParams.imageHeight;
  if (!(pixelX < imageWidth && pixelY < imageHeight))
    return;

  int gridWidth = imageWidth / squareSize;
  int squareX = pixelX / squareSize;
  int squareY = pixelY / squareSize;
  int squareIndex = squareY * gridWidth + squareX;
  int count = numCircles[squareIndex];
  int offset = indexOffsets[squareIndex];

  for (int i = 0; i < count; i++) {
    int index = circleIndices[offset + i];
    int index3 = 3 * index;

    // read position
    float3 p = *(float3 *)(&cuConstRendererParams.position[index3]);

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    float4 *imgPtr =
        (float4 *)(&cuConstRendererParams
                        .imageData[4 * (pixelY * imageWidth + pixelX)]);
    float2 pixelCenterNorm =
        make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                    invHeight * (static_cast<float>(pixelY) + 0.5f));
    shadePixel(index, pixelCenterNorm, p, imgPtr);
  }
}

__global__ void kernelRange(int length, int *circleIndices) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < length)
    circleIndices[index] = index;
}

struct CircleInBoxPredicate {
  float boxL;
  float boxR;
  float boxT;
  float boxB;

  __device__ bool operator()(int circleIndex) {
    float3 p = *(float3 *)(&cuConstRendererParams.position[3 * circleIndex]);
    float rad = cuConstRendererParams.radius[circleIndex];
    return circleInBoxConservative(p.x, p.y, rad, boxL, boxR, boxT, boxB);
  }
};

////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer() {
  image = NULL;

  numCircles = 0;
  position = NULL;
  velocity = NULL;
  color = NULL;
  radius = NULL;

  cudaDevicePosition = NULL;
  cudaDeviceVelocity = NULL;
  cudaDeviceColor = NULL;
  cudaDeviceRadius = NULL;
  cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

  if (image) {
    delete image;
  }

  if (position) {
    delete[] position;
    delete[] velocity;
    delete[] color;
    delete[] radius;
  }

  if (cudaDevicePosition) {
    hipFree(cudaDevicePosition);
    hipFree(cudaDeviceVelocity);
    hipFree(cudaDeviceColor);
    hipFree(cudaDeviceRadius);
    hipFree(cudaDeviceImageData);
  }
}

const Image *CudaRenderer::getImage() {

  // need to copy contents of the rendered image from device memory
  // before we expose the Image object to the caller

  printf("Copying image data from device\n");

  hipMemcpy(image->data, cudaDeviceImageData,
             sizeof(float) * 4 * image->width * image->height,
             hipMemcpyDeviceToHost);

  return image;
}

void CudaRenderer::loadScene(SceneName scene) {
  sceneName = scene;
  loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void CudaRenderer::setup() {

  int deviceCount = 0;
  std::string name;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Initializing CUDA for CudaRenderer\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    name = deviceProps.name;

    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");

  // By this time the scene should be loaded.  Now copy all the key
  // data structures into device memory so they are accessible to
  // CUDA kernels
  //
  // See the CUDA Programmer's Guide for descriptions of
  // hipMalloc and hipMemcpy

  hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
  hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
  hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
  hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
  hipMalloc(&cudaDeviceImageData,
             sizeof(float) * 4 * image->width * image->height);

  hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles,
             hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles,
             hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles,
             hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles,
             hipMemcpyHostToDevice);

  // Initialize parameters in constant memory.  We didn't talk about
  // constant memory in class, but the use of read-only constant
  // memory here is an optimization over just sticking these values
  // in device global memory.  NVIDIA GPUs have a few special tricks
  // for optimizing access to constant memory.  Using global memory
  // here would have worked just as well.  See the Programmer's
  // Guide for more information about constant memory.

  GlobalConstants params;
  params.sceneName = sceneName;
  params.numCircles = numCircles;
  params.imageWidth = image->width;
  params.imageHeight = image->height;
  params.position = cudaDevicePosition;
  params.velocity = cudaDeviceVelocity;
  params.color = cudaDeviceColor;
  params.radius = cudaDeviceRadius;
  params.imageData = cudaDeviceImageData;

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

  // also need to copy over the noise lookup tables, so we can
  // implement noise on the GPU
  int *permX;
  int *permY;
  float *value1D;
  getNoiseTables(&permX, &permY, &value1D);
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

  // last, copy over the color table that's used by the shading
  // function for circles in the snowflake demo

  float lookupTable[COLOR_MAP_SIZE][3] = {
      {1.f, 1.f, 1.f}, {1.f, 1.f, 1.f},  {.8f, .9f, 1.f},
      {.8f, .9f, 1.f}, {.8f, 0.8f, 1.f},
  };

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable,
                     sizeof(float) * 3 * COLOR_MAP_SIZE);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void CudaRenderer::allocOutputImage(int width, int height) {

  if (image)
    delete image;
  image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void CudaRenderer::clearImage() {

  // 256 threads per block is a healthy number
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x,
               (image->height + blockDim.y - 1) / blockDim.y);

  if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
    kernelClearImageSnowflake<<<gridDim, blockDim>>>();
  } else {
    kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
  }
  hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void CudaRenderer::advanceAnimation() {
  // 256 threads per block is a healthy number
  dim3 blockDim(256, 1);
  dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

  // only the snowflake scene has animation
  if (sceneName == SNOWFLAKES) {
    kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
  } else if (sceneName == BOUNCING_BALLS) {
    kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
  } else if (sceneName == HYPNOSIS) {
    kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
  } else if (sceneName == FIREWORKS) {
    kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
  }
  hipDeviceSynchronize();
}

void CudaRenderer::render() {
  int imageSize = 1024;
  if (!(image->width == imageSize && image->height == imageSize)) {
    printf("This implementation assumes that the image is %dx%d.", imageSize,
           imageSize);
    exit(1);
  }
  float invSize = 1.f / imageSize;

  int threadsPerBlock = 256;

  // During each iteration, we consider the image to be divided into squares of
  // a certain side length.
  int squareSize = imageSize;

  // For each square, we want to keep a list of the indices of all circles that
  // potentially overlap with that square (not necessarily, e.g. if we're using
  // a conservative check). At the beginning, there's just one square (the
  // entire image), so we just consider all circles to potentially overlap it.
  // This `circleIndexLengths` vector keeps track of the number of circles
  // overlapping with each square.
  std::vector<int> circleIndexLengths = {numCircles};

  // In each iteration, we put all these lists of indices into a single
  // `circleIndices` allocation, and we use a second `circleIndexIndices` vector
  // to track the start index in `circleIndices` for the list for each square.
  std::vector<int> circleIndexIndices = {0};
  int *circleIndices;
  cudaCheckError(hipMalloc(&circleIndices, sizeof(int) * numCircles));

  // At the start, we need to set the circle indices to just be the indices of
  // all the circles.
  int numBlocks = (numCircles + threadsPerBlock - 1) / threadsPerBlock;
  kernelRange<<<numBlocks, threadsPerBlock>>>(numCircles, circleIndices);
  cudaCheckError(hipDeviceSynchronize());

  // We also track the sum of the number of circles in each list across all
  // squares, so that in each iteration we can just multiply this number by the
  // square count ratio to get the size of the next allocation.
  int totalCirclesAcrossSquares = numCircles;

  // We work our way down to smaller and smaller squares.
  std::vector<int> squareSizes = {512, 256, 128, 64, 32, 16};
  for (int nextSquareSize : squareSizes) {
    // Conservatively allocate enough memory to store all circle lists even if
    // every subsquare overlaps with all the same circles as its parent square.
    int ratio = squareSize / nextSquareSize;
    int *nextCircleIndices;
    size_t bytes = sizeof(int) * (ratio * ratio) * totalCirclesAcrossSquares;
    cudaCheckError(hipMalloc(&nextCircleIndices, bytes));
    totalCirclesAcrossSquares = 0;

    // Allocate length and offset arrays for the new iteration.
    int nextGridSize = imageSize / nextSquareSize;
    std::vector<int> nextCircleIndexLengths(nextGridSize * nextGridSize);
    std::vector<int> nextCircleIndexIndices(nextGridSize * nextGridSize);

    // Iterate over all the larger squares.
    int gridSize = imageSize / squareSize;
    int nextCircleIndexIndex = 0;
    for (int squareY = 0; squareY < gridSize; squareY++) {
      for (int squareX = 0; squareX < gridSize; squareX++) {
        // Each larger square has a list of circles that may overlap with it;
        // we'll filter this list for each smaller square in this square.
        int squareIndex = squareY * gridSize + squareX;
        int circleIndexLength = circleIndexLengths[squareIndex];
        int circleIndexIndex = circleIndexIndices[squareIndex];

        // Now iterate over all the smaller squares within this larger one.
        int childIndex = 0;
        for (int childSquareY = 0; childSquareY < ratio; childSquareY++) {
          int nextSquareY = squareY * ratio + childSquareY;
          for (int childSquareX = 0; childSquareX < ratio; childSquareX++) {
            int nextSquareX = squareX * ratio + childSquareX;

            int nextSquareIndex = nextSquareY * nextGridSize + nextSquareX;
            nextCircleIndexIndices[nextSquareIndex] = nextCircleIndexIndex;

            int *input = circleIndices + circleIndexIndex;
            int *output = nextCircleIndices + nextCircleIndexIndex;

            float boxL =
                invSize * static_cast<float>(nextSquareX * nextSquareSize);
            float boxR = invSize *
                         static_cast<float>((nextSquareX + 1) * nextSquareSize);
            float boxT = invSize *
                         static_cast<float>((nextSquareY + 1) * nextSquareSize);
            float boxB =
                invSize * static_cast<float>(nextSquareY * nextSquareSize);
            CircleInBoxPredicate predicate{boxL, boxR, boxT, boxB};

            thrust::device_ptr<int> outputEnd = thrust::copy_if(
                thrust::device_pointer_cast(input),
                thrust::device_pointer_cast(input + circleIndexLength),
                thrust::device_pointer_cast(output), predicate);
            cudaCheckError(hipDeviceSynchronize());

            int nextCircleIndexLength = outputEnd.get() - output;
            nextCircleIndexLengths[nextSquareIndex] = nextCircleIndexLength;
            totalCirclesAcrossSquares += nextCircleIndexLength;

            childIndex++;
            nextCircleIndexIndex += circleIndexLength;
          }
        }
      }
    }

    // This next iteration has now become the current one.
    cudaCheckError(hipFree(circleIndices));
    circleIndices = nextCircleIndices;
    circleIndexIndices = nextCircleIndexIndices;
    circleIndexLengths = nextCircleIndexLengths;
    squareSize = nextSquareSize;
  }

  int numSquares = (imageSize / squareSize) * (imageSize / squareSize);

  int *numCircles;
  int *indexOffsets;

  cudaCheckError(hipMalloc(&numCircles, sizeof(int) * numSquares));
  cudaCheckError(hipMalloc(&indexOffsets, sizeof(int) * numSquares));

  cudaCheckError(hipMemcpy(numCircles, circleIndexLengths.data(),
                            sizeof(int) * numSquares, hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(indexOffsets, circleIndexIndices.data(),
                            sizeof(int) * numSquares, hipMemcpyHostToDevice));

  // 256 threads per block is a healthy number
  dim3 blockDim(16, 16);
  dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x,
               (image->height + blockDim.y - 1) / blockDim.y);

  kernelRenderCircles<<<gridDim, blockDim>>>(squareSize, numCircles,
                                             indexOffsets, circleIndices);
  cudaCheckError(hipDeviceSynchronize());

  cudaCheckError(hipFree(indexOffsets));
  cudaCheckError(hipFree(numCircles));
}
